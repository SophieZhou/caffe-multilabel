#include "hip/hip_runtime.h"
// Copyright 2014 BVLC and contributors.

#include <algorithm>
#include <cmath>
#include <cstdlib>
#include <cstring>

#include "caffe/common.hpp"
#include "caffe/util/im2col.hpp"

namespace caffe {

	template <typename Dtype>
__global__ void bu_im2col_gpu_kernel(
	const int n, const Dtype* data_im,
	const int height, const int width, const int ksize, const int pad,
	const int stride, const int height_col, const int width_col,
	Dtype* data_col,
	const int data_im_size,
	const int data_col_size,
	const int batch_size,
	const bool trans) 
{
	/*for(int batch_index = 0; batch_index < batch_size; batch_index++)
	{
		for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < n; index += blockDim.x * gridDim.x){
			int w_out = index % width_col;
			int h_index = index / width_col;
			int h_out = h_index % height_col;
			int channel_in = h_index / height_col;
			int channel_out = channel_in * ksize * ksize;
			int h_in = h_out * stride - pad;
			int w_in = w_out * stride - pad;
			Dtype* data_col_ptr = data_col;
			data_col_ptr += batch_index* data_col_size + (channel_out * height_col + h_out) * width_col + w_out;
			const Dtype* data_im_ptr = data_im;
			data_im_ptr += batch_index* data_im_size + (channel_in * height + h_in) * width + w_in;

			for (int i = 0; i < ksize; ++i) {
				for (int j = 0; j < ksize; ++j) {
					int h = h_in + i;
					int w = w_in + j;
					*data_col_ptr = (h >= 0 && w >= 0 && h < height && w < width) ?
						data_im_ptr[i * width + j]  : 0;
					data_col_ptr += height_col * width_col;
				}
			}

		}
	}*/
	int N = height_col * width_col;
	int channels_all = n / width_col / height_col;
	int trans_col_offset = ksize * ksize * channels_all;

	for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < n; index += blockDim.x * gridDim.x){
		int w_out = index % width_col;
		int h_index = index / width_col;
		int h_out = h_index % height_col;
		int channel_in = h_index / height_col;
		int channel_out = channel_in * ksize * ksize;
		int h_in = h_out * stride - pad;
		int w_in = w_out * stride - pad;

		if (!trans){
			Dtype* data_col_ptr = data_col;
			data_col_ptr += channel_out * N * batch_size + h_out * width_col + w_out;

			const Dtype* data_im_ptr = data_im;
			data_im_ptr += (channel_in * height + h_in) * width + w_in;


			for(int batch_index = 0; batch_index < batch_size; batch_index++)
			{
				Dtype* data_write_col_ptr = data_col_ptr;

				for (int i = 0; i < ksize; ++i) {
					for (int j = 0; j < ksize; ++j) {
						int h = h_in + i;
						int w = w_in + j;
						*data_write_col_ptr = (h >= 0 && w >= 0 && h < height && w < width) ?
							data_im_ptr[i * width + j]  : 0;
						data_write_col_ptr += N * batch_size;
					}
				}

				data_col_ptr += N;
				data_im_ptr += data_im_size;
			}
		}else{	

			Dtype* data_col_ptr = data_col;
			data_col_ptr += (h_out * width_col + w_out) * batch_size * trans_col_offset + channel_in * ksize * ksize;
				//index * batch_size * ksize * ksize;

			const Dtype* data_im_ptr = data_im;
			data_im_ptr += (channel_in * height + h_in) * width + w_in;

			for(int batch_index = 0; batch_index < batch_size; batch_index++)
			{
				Dtype* data_write_col_ptr = data_col_ptr;

				for (int i = 0; i < ksize; ++i) {
					for (int j = 0; j < ksize; ++j) {
						int h = h_in + i;
						int w = w_in + j;
						*data_write_col_ptr = (h >= 0 && w >= 0 && h < height && w < width) ?
							data_im_ptr[i * width + j]  : 0;
						data_write_col_ptr ++;
					}
				}

				data_col_ptr += trans_col_offset;
				data_im_ptr += data_im_size;
			}
		}
	}
}


template <typename Dtype>
void bu_im2col_gpu(const Dtype* data_im, const int channels,
				   const int height, const int width, const int ksize, const int pad,
				   const int stride, Dtype* data_col, const int batch_size, const bool trans){
	// We are going to launch channels * height_col * width_col kernels, each
	// kernel responsible for copying a single-channel grid.
	int height_col = (height + 2 * pad - ksize) / stride + 1;
	int width_col = (width + 2 * pad - ksize) / stride + 1;
	int num_kernels = channels * height_col * width_col;

	int data_im_size = height*width*channels;
	int data_col_size = num_kernels*ksize*ksize;

	// NOLINT_NEXT_LINE(whitespace/operators)
	bu_im2col_gpu_kernel<Dtype><<<CAFFE_GET_BLOCKS(num_kernels), // num_kernels/16, means each thread process 16 elements
		CAFFE_CUDA_NUM_THREADS, 0, Caffe::get_current_cu_stream()>>>(
		num_kernels, data_im, height, width, ksize, pad, stride, height_col,
		width_col, data_col, data_im_size, data_col_size, batch_size, trans);
	CUDA_POST_KERNEL_CHECK;
}

// Explicit instantiation
template void bu_im2col_gpu<float>(
	const float* data_im, const int channels,
	const int height, const int width, const int ksize, const int pad,
	const int stride, float* data_col,
	const int batch_size, const bool trans);
template void bu_im2col_gpu<double>(
	const double* data_im, const int channels,
	const int height, const int width, const int ksize, const int pad,
	const int stride, double* data_col,
	const int batch_size, const bool trans);

template <typename Dtype>
void bu_im2col_gpu(const Dtype* data_im, const int channels,
				   const int height, const int width, const int ksize, const int pad,
				   const int stride, Dtype* data_col, const int batch_size)
{
	bu_im2col_gpu(data_im, channels,
					height, width, ksize, pad,
					stride, data_col, batch_size, false);
}





// Explicit instantiation
template void bu_im2col_gpu<float>(
	const float* data_im, const int channels,
	const int height, const int width, const int ksize, const int pad,
	const int stride, float* data_col,
	const int batch_size);
template void bu_im2col_gpu<double>(
	const double* data_im, const int channels,
	const int height, const int width, const int ksize, const int pad,
	const int stride, double* data_col,
	const int batch_size);


template <typename Dtype>
__global__ void bu_im2col_gpu_kernel_rot(
	const int n, const Dtype* data_im,
	const int height, const int width, const int ksize, const int pad,
	const int stride, const int height_col, const int width_col,
	Dtype* data_col,
	const int data_im_size,
	const int data_col_size,
	const int batch_size) 
{
	for(int batch_index = 0; batch_index < batch_size; batch_index++)
	{
		for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < n; index += blockDim.x * gridDim.x){
			int w_out = index % width_col;
			int h_index = index / width_col;
			int h_out = h_index % height_col;
			int channel_in = h_index / height_col;
			int channel_out = channel_in * ksize * ksize;
			int h_in = h_out * stride - pad;
			int w_in = w_out * stride - pad;
			Dtype* data_col_ptr = data_col;
			data_col_ptr += batch_index* data_col_size + (channel_out * height_col + h_out) * width_col + w_out;
			const Dtype* data_im_ptr = data_im;
			data_im_ptr += batch_index* data_im_size + (channel_in * height + h_in) * width + w_in;

			for (int i = 0; i < ksize; ++i) {
				for (int j = 0; j < ksize; ++j) {
					int h = h_in + i;
					int w = w_in + j;
					*data_col_ptr = (h >= 0 && w >= 0 && h < height && w < width) ?
						data_im_ptr[i * width + j]  : 0;
					data_col_ptr += height_col * width_col;
				}
			}

		}
	}
}

template <typename Dtype>
void bu_im2col_gpu_rot(const Dtype* data_im, const int channels,
				   const int height, const int width, const int ksize, const int pad,
				   const int stride, Dtype* data_col, const int batch_size)
{
	// We are going to launch channels * height_col * width_col kernels, each
	// kernel responsible for copying a single-channel grid.
	int height_col = (height + 2 * pad - ksize) / stride + 1;
	int width_col = (width + 2 * pad - ksize) / stride + 1;
	int num_kernels = channels * height_col * width_col;

	int data_im_size = height*width*channels;
	int data_col_size = num_kernels*ksize*ksize;
	// NOLINT_NEXT_LINE(whitespace/operators)
	bu_im2col_gpu_kernel_rot<Dtype><<<CAFFE_GET_BLOCKS(num_kernels), // num_kernels/16, means each thread process 16 elements
		CAFFE_CUDA_NUM_THREADS, 0, Caffe::get_current_cu_stream()>>>(
		num_kernels, data_im, height, width, ksize, pad, stride, height_col,
		width_col, data_col, data_im_size, data_col_size, batch_size);
	CUDA_POST_KERNEL_CHECK;
}

// Explicit instantiation
template void bu_im2col_gpu_rot<float>(
	const float* data_im, const int channels,
	const int height, const int width, const int ksize, const int pad,
	const int stride, float* data_col,
	const int batch_size);
template void bu_im2col_gpu_rot<double>(
	const double* data_im, const int channels,
	const int height, const int width, const int ksize, const int pad,
	const int stride, double* data_col,
	const int batch_size);


template <typename Dtype>
__global__ void im2col_gpu_kernel(const int n, const Dtype* data_im,
    const int height, const int width, const int ksize, const int pad,
    const int stride, const int height_col, const int width_col,
    Dtype* data_col) {
  CUDA_KERNEL_LOOP(index, n) {
    int w_out = index % width_col;
    index /= width_col;
    int h_out = index % height_col;
    int channel_in = index / height_col;
    int channel_out = channel_in * ksize * ksize;
    int h_in = h_out * stride - pad;
    int w_in = w_out * stride - pad;
    data_col += (channel_out * height_col + h_out) * width_col + w_out;
    data_im += (channel_in * height + h_in) * width + w_in;
    for (int i = 0; i < ksize; ++i) {
      for (int j = 0; j < ksize; ++j) {
        int h = h_in
			+ i;
        int w = w_in + j;
        *data_col = (h >= 0 && w >= 0 && h < height && w < width) ?
            data_im[i * width + j] : 0;
        data_col += height_col * width_col;
      }
    }
  }
}

template <typename Dtype>
void im2col_gpu(const Dtype* data_im, const int channels,
    const int height, const int width, const int ksize, const int pad,
    const int stride, Dtype* data_col) {
  // We are going to launch channels * height_col * width_col kernels, each
  // kernel responsible for copying a single-channel grid.
  int height_col = (height + 2 * pad - ksize) / stride + 1;
  int width_col = (width + 2 * pad - ksize) / stride + 1;
  int num_kernels = channels * height_col * width_col;
  // NOLINT_NEXT_LINE(whitespace/operators)
  im2col_gpu_kernel<Dtype><<<CAFFE_GET_BLOCKS(num_kernels),
                             CAFFE_CUDA_NUM_THREADS>>>(
      num_kernels, data_im, height, width, ksize, pad, stride, height_col,
      width_col, data_col);
  CUDA_POST_KERNEL_CHECK;
}


// Explicit instantiation
template void im2col_gpu<float>(const float* data_im, const int channels,
    const int height, const int width, const int ksize, const int pad,
    const int stride, float* data_col);
template void im2col_gpu<double>(const double* data_im, const int channels,
    const int height, const int width, const int ksize, const int pad,
    const int stride, double* data_col);

template <typename Dtype>
__global__ void col2im_gpu_kernel(const int n, const Dtype* data_col,
    const int height, const int width, const int channels, const int ksize,
    const int pad, const int stride, const int height_col, const int width_col,
    Dtype* data_im) {
  CUDA_KERNEL_LOOP(index, n) {
    Dtype val = 0;
    int w = index % width + pad;
    int h = (index / width) % height + pad;
    int c = index / (width * height);
    // compute the start and end of the output
    int w_col_start = (w < ksize) ? 0 : (w - ksize) / stride + 1;
    int w_col_end = min(w / stride + 1, width_col);
    int h_col_start = (h < ksize) ? 0 : (h - ksize) / stride + 1;
    int h_col_end = min(h / stride + 1, height_col);
    /*
    for (int h_col = h_col_start; h_col < h_col_end; ++h_col) {
      for (int w_col = w_col_start; w_col < w_col_end; ++w_col) {
        // the col location: [c * width * height + h_out, w_out]
        int c_col = c * ksize * ksize + (h - h_col * stride) * ksize + (w - w_col * stride);
        val += data_col[(c_col * height_col + h_col) * width_col + w_col];
      }
    }
    */
    // equivalent implementation
    int offset = (c * ksize * ksize + h * ksize + w) * height_col * width_col;
    int coeff_h_col = (1 - stride * ksize * height_col) * width_col;
    int coeff_w_col = (1 - stride * height_col * width_col);
    for (int h_col = h_col_start; h_col < h_col_end; ++h_col) {
      for (int w_col = w_col_start; w_col < w_col_end; ++w_col) {
        val += data_col[offset + h_col * coeff_h_col + w_col * coeff_w_col];
      }
    }
    data_im[index] = val;
  }
}

template <typename Dtype>
void col2im_gpu(const Dtype* data_col, const int channels,
    const int height, const int width, const int ksize, const int pad,
    const int stride, Dtype* data_im) {
  // CUDA_CHECK(hipMemset(data_im, 0,
  //            sizeof(Dtype) * height * width * channels));
  int height_col = (height + 2 * pad - ksize) / stride + 1;
  int width_col = (width + 2 * pad - ksize) / stride + 1;
  int num_kernels = channels * height * width;
  // To avoid involving atomic operations, we will launch one kernel per
  // bottom dimension, and then in the kernel add up the top dimensions.
  // NOLINT_NEXT_LINE(whitespace/operators)
  col2im_gpu_kernel<Dtype><<<CAFFE_GET_BLOCKS(num_kernels),
                             CAFFE_CUDA_NUM_THREADS>>>(
      num_kernels, data_col, height, width, channels, ksize, pad, stride,
      height_col, width_col, data_im);
  CUDA_POST_KERNEL_CHECK;
}


// Explicit instantiation
template void col2im_gpu<float>(const float* data_col, const int channels,
    const int height, const int width, const int psize, const int pad,
    const int stride, float* data_im);
template void col2im_gpu<double>(const double* data_col, const int channels,
    const int height, const int width, const int psize, const int pad,
    const int stride, double* data_im);



//Enable batched col2im
template <typename Dtype>
__global__ void bu_col2im_gpu_kernel(const int n, const Dtype* data_col,
    const int height, const int width, const int channels, const int ksize,
    const int pad, const int stride, const int height_col, const int width_col,
    Dtype* data_im,
	const int batch_size, bool trans) 
{
	CUDA_KERNEL_LOOP(index, n) 
	{

		//
		int col_length = channels*ksize*ksize;
		int col_offset = height_col*width_col; // offset per col image
		int im_offset = n;
		int t_index = index;
		int col_start = 0;

		int w = index % width + pad;
		int h = (index / width) % height + pad;
		int c = index / (width * height);
		// compute the start and end of the output
		int w_col_start = (w < ksize) ? 0 : (w - ksize) / stride + 1;
		int w_col_end = min(w / stride + 1, width_col);
		int h_col_start = (h < ksize) ? 0 : (h - ksize) / stride + 1;
		int h_col_end = min(h / stride + 1, height_col);

		if (!trans)
		{
			for (int batch_index = 0; batch_index<batch_size; batch_index++)
			{
				Dtype val = 0;
				/*
				for (int h_col = h_col_start; h_col < h_col_end; ++h_col) {
				for (int w_col = w_col_start; w_col < w_col_end; ++w_col) {
				// the col location: [c * width * height + h_out, w_out]
				int c_col = c * ksize * ksize + (h - h_col * stride) * ksize + (w - w_col * stride);
				val += data_col[(c_col * height_col + h_col) * width_col + w_col];
				}
				}
				*/
				// equivalent implementation


				int offset = (c * ksize * ksize + h * ksize + w) * height_col * width_col+col_start;
				int coeff_h_col = (1 - stride * ksize * height_col) * width_col;
				int coeff_w_col = (1 - stride * height_col * width_col);
				for (int h_col = h_col_start; h_col < h_col_end; ++h_col) 
				{
					for (int w_col = w_col_start; w_col < w_col_end; ++w_col) 
					{
						val += data_col[offset + h_col * coeff_h_col + w_col * coeff_w_col];
					}
				}
				data_im[t_index] = val;
				t_index += n;
				col_start += col_offset;
			}
		}
		else
		{
			for (int batch_index = 0; batch_index < batch_size; batch_index++)
			{
				Dtype val = 0;
			//	for (int h_col = h_col_start; h_col < h_col_end; ++h_col)
			//	{
			//		for (int w_col = w_col_start; w_col < w_col_end; ++w_col)
			//		{
			//			int N_index = weight_col * h_col + w_col;
			//			int k_index = c * ksize * ksize + (h - h_col * stride)* ksize + w - stride * w_col;

			//			int col_index = batch_index * col_offset + N_index * batch_size * col_offset + k_index;
			//			val += data_col[col_index];
			//		}
			//	}

				int offset = col_start + (c * ksize *ksize + h * ksize + w)* batch_size;
				int coeff_h_col = (width_col  - ksize  * stride) * batch_size * col_length;
				int coeff_w_col = (col_length -  stride) * batch_size;
				for (int h_col = h_col_start; h_col < h_col_end; ++h_col)
				{
					for (int w_col = w_col_start; w_col < w_col_end; ++w_col)
					{
						val += data_col[offset + coeff_h_col * h_col + coeff_w_col * w_col];
					}
				}

				data_im[t_index] = val;
				t_index += n;
				col_start += 1;
			}

		}
	}
}

template <typename Dtype>
void bu_col2im_gpu(const Dtype* data_col, const int channels,
    const int height, const int width, const int ksize, const int pad,
    const int stride, Dtype* data_im,
	const int batch_size) {
  // CUDA_CHECK(hipMemset(data_im, 0,
  //            sizeof(Dtype) * height * width * channels));
  int height_col = (height + 2 * pad - ksize) / stride + 1;
  int width_col = (width + 2 * pad - ksize) / stride + 1;
  int num_kernels = channels * height * width;
  // To avoid involving atomic operations, we will launch one kernel per
  // bottom dimension, and then in the kernel add up the top dimensions.
  // NOLINT_NEXT_LINE(whitespace/operators)
  bu_col2im_gpu_kernel<Dtype><<<CAFFE_GET_BLOCKS(num_kernels),
                             CAFFE_CUDA_NUM_THREADS, 0, Caffe::get_current_cu_stream() >>>(
      num_kernels, data_col, height, width, channels, ksize, pad, stride,
      height_col, width_col, data_im,
	  batch_size, false);
  CUDA_POST_KERNEL_CHECK;
}


// Explicit instantiation
template void bu_col2im_gpu<float>(const float* data_col, const int channels,
    const int height, const int width, const int psize, const int pad,
    const int stride, float* data_im,
	const int batch_size);
template void bu_col2im_gpu<double>(const double* data_col, const int channels,
    const int height, const int width, const int psize, const int pad,
    const int stride, double* data_im,
	const int batch_size);

template <typename Dtype>
void bu_col2im_gpu(const Dtype* data_col, const int channels,
    const int height, const int width, const int ksize, const int pad,
    const int stride, Dtype* data_im,
	const int batch_size, bool trans) {
  // CUDA_CHECK(hipMemset(data_im, 0,
  //            sizeof(Dtype) * height * width * channels));
  int height_col = (height + 2 * pad - ksize) / stride + 1;
  int width_col = (width + 2 * pad - ksize) / stride + 1;
  int num_kernels = channels * height * width;
  // To avoid involving atomic operations, we will launch one kernel per
  // bottom dimension, and then in the kernel add up the top dimensions.
  // NOLINT_NEXT_LINE(whitespace/operators)
  bu_col2im_gpu_kernel<Dtype><<<CAFFE_GET_BLOCKS(num_kernels),
                             CAFFE_CUDA_NUM_THREADS, 0, Caffe::get_current_cu_stream() >>>(
      num_kernels, data_col, height, width, channels, ksize, pad, stride,
      height_col, width_col, data_im,
	  batch_size, trans);
  CUDA_POST_KERNEL_CHECK;
}

template void bu_col2im_gpu<float>(const float* data_col, const int channels,
    const int height, const int width, const int psize, const int pad,
    const int stride, float* data_im,
	const int batch_size, bool trans);
template void bu_col2im_gpu<double>(const double* data_col, const int channels,
    const int height, const int width, const int psize, const int pad,
    const int stride, double* data_im,
	const int batch_size, bool trans);


//Enable batched col2im
template <typename Dtype>
__global__ void bu_col2im_gpu_rot_kernel(const int n, const Dtype* data_col,
    const int height, const int width, const int channels, const int ksize,
    const int pad, const int stride, const int height_col, const int width_col,
    Dtype* data_im,
	const int batch_size) {
  CUDA_KERNEL_LOOP(index, n) {
    
	//
	int col_length = channels*ksize*ksize;
	int col_offset = height_col*width_col; // offset per col image
	int im_offset = n;
	int t_index = index;

    int w = index % width + pad;
    int h = (index / width) % height + pad;
    int c = index / (width * height);
    // compute the start and end of the output
    int w_col_start = (w < ksize) ? 0 : (w - ksize) / stride + 1;
    int w_col_end = min(w / stride + 1, width_col);
    int h_col_start = (h < ksize) ? 0 : (h - ksize) / stride + 1;
    int h_col_end = min(h / stride + 1, height_col);

    // every batch, offset height_col * width_col
	int offset = (c * ksize * ksize + h * ksize + w) * height_col * width_col*batch_size;
	int coeff_h_col = (1 - stride * ksize * height_col*batch_size) * width_col;
	int coeff_w_col = (1 - stride * height_col * width_col*batch_size);
	for (int batch_index = 0; batch_index<batch_size; batch_index++){
		Dtype val = 0;
		
		for (int h_col = h_col_start; h_col < h_col_end; ++h_col) {
			for (int w_col = w_col_start; w_col < w_col_end; ++w_col) {
				val += data_col[offset + h_col * coeff_h_col + w_col * coeff_w_col];
			}
		}
		data_im[t_index] = val;
		t_index += n;
		offset += col_offset;
	}
  }
}

template <typename Dtype>
void bu_col2im_gpu_rot(const Dtype* data_col, const int channels,
    const int height, const int width, const int ksize, const int pad,
    const int stride, Dtype* data_im,
	const int batch_size) {
  // CUDA_CHECK(hipMemset(data_im, 0,
  //            sizeof(Dtype) * height * width * channels));
  int height_col = (height + 2 * pad - ksize) / stride + 1;
  int width_col = (width + 2 * pad - ksize) / stride + 1;
  int num_kernels = channels * height * width;
  // To avoid involving atomic operations, we will launch one kernel per
  // bottom dimension, and then in the kernel add up the top dimensions.
  // NOLINT_NEXT_LINE(whitespace/operators)
  bu_col2im_gpu_rot_kernel<Dtype><<<CAFFE_GET_BLOCKS(num_kernels),
                             CAFFE_CUDA_NUM_THREADS, 0, Caffe::get_current_cu_stream()>>>(
      num_kernels, data_col, height, width, channels, ksize, pad, stride,
      height_col, width_col, data_im,
	  batch_size);
  CUDA_POST_KERNEL_CHECK;
}


// Explicit instantiation
template void bu_col2im_gpu_rot<float>(const float* data_col, const int channels,
    const int height, const int width, const int psize, const int pad,
    const int stride, float* data_im,
	const int batch_size);
template void bu_col2im_gpu_rot<double>(const double* data_col, const int channels,
    const int height, const int width, const int psize, const int pad,
    const int stride, double* data_im,
	const int batch_size);

//Composite contigious images to one row-major matrix (long row)
template <typename Dtype>
__global__ void cu_im2mat_gpu_kernel(const int n, const Dtype* data_im,
    const int height, const int width, const int channels,
    Dtype* data_mat,
	const int batch_size) {
  CUDA_KERNEL_LOOP(index, n) {
    int temp = index;
	int col_now = temp % width;
	temp /= width;
	int row_now = temp % height;
	int ch_now = temp / height;

	int im_offset = n;
	int mat_offset = width;

	const Dtype *im_ptr = data_im + index;
	Dtype *mat_ptr = data_mat + col_now + (row_now + ch_now * height) * width * batch_size;


	for (int batch_index = 0; batch_index < batch_size; batch_index++){
		*(mat_ptr) = *(im_ptr);

		//offset both indexers
		im_ptr += im_offset;
		mat_ptr += mat_offset;
	}

	
  }
}

template <typename Dtype>
void cu_im2mat_gpu(const Dtype* data_im, const int channels,
    const int height, const int width, 
	Dtype* data_mat,
	const int batch_size) {
  // CUDA_CHECK(hipMemset(data_im, 0,
  //            sizeof(Dtype) * height * width * channels));
  int num_kernels = channels * height * width;
  // To avoid involving atomic operations, we will launch one kernel per
  // bottom dimension, and then in the kernel add up the top dimensions.
  // NOLINT_NEXT_LINE(whitespace/operators)
  cu_im2mat_gpu_kernel<Dtype><<<CAFFE_GET_BLOCKS(num_kernels),
                             CAFFE_CUDA_NUM_THREADS, 0, Caffe::get_current_cu_stream()>>>(
      num_kernels, data_im, height, width, channels, 
	  data_mat,
	  batch_size);
  CUDA_POST_KERNEL_CHECK;
}


// Explicit instantiation
template void cu_im2mat_gpu<float>(const float* data_im, const int channels,
    const int height, const int width, 
	float* data_mat,
	const int batch_size);
template void cu_im2mat_gpu<double>(const double* data_im, const int channels,
    const int height, const int width, 
	double* data_mat,
	const int batch_size);


//Convert composited big matrix (long row) to row-major images with carry
//Carry is added by A + beta * C
template <typename Dtype>
__global__ void cu_mat2im_c_gpu_kernel(const int n, const Dtype* data_mat,
    const int mat_height, const int mat_width,
    Dtype* data_im,
	Dtype beta,
	Dtype* data_carry,
	const int batch_size) {
  CUDA_KERNEL_LOOP(index, n) {
	int col_now = index % mat_width;
	int row_now = index / mat_width;

	int im_idx = index;
	int mat_idx = col_now + row_now * mat_width * batch_size;

	Dtype carry = data_carry[index] * beta;

	int im_offset = n;
	int mat_offset = mat_width;
	
	for (int batch_index = 0; batch_index < batch_size; batch_index++){

		//memory reordering and carry adding
		data_im[im_idx] = data_mat[mat_idx]+carry;
		

		//offset both indexers
		im_idx += im_offset;
		mat_idx += mat_offset;
	}
	
  }
}

template <typename Dtype>
void cu_mat2im_c_gpu(const Dtype* data_mat,
    const int mat_height, const int mat_width, 
	Dtype* data_im,
	Dtype beta,
	Dtype* data_carry,
	const int batch_size) {
  // CUDA_CHECK(hipMemset(data_im, 0,
  //            sizeof(Dtype) * height * width * channels));
  int num_kernels = mat_height * mat_width;
  // To avoid involving atomic operations, we will launch one kernel per
  // bottom dimension, and then in the kernel add up the top dimensions.
  // NOLINT_NEXT_LINE(whitespace/operators)
  cu_mat2im_c_gpu_kernel<Dtype><<<CAFFE_GET_BLOCKS(num_kernels),
                             CAFFE_CUDA_NUM_THREADS, 0, Caffe::get_current_cu_stream()>>>(
      num_kernels, data_mat, mat_height, mat_width,
	  data_im,
	  beta,
	  data_carry,
	  batch_size);
  //CUDA_POST_KERNEL_CHECK;
}


// Explicit instantiation
template void cu_mat2im_c_gpu<float>(const float* data_mat,
    const int mat_height, const int mat_width, 
	float* data_im,
	float beta,
	float* data_carry,
	const int batch_size);
template void cu_mat2im_c_gpu<double>(const double* data_mat,
    const int mat_height, const int mat_width, 
	double* data_im,
	double beta,
	double* data_carry,
	const int batch_size);


template <typename Dtype>
__global__ void cu_trans_3D_acc_gpu_kernel(const int n, const Dtype* data_src, Dtype *data_dst,
    const int X, const int Y, const int Z, int mX, int mY, int mZ, Dtype alpha)
{
	CUDA_KERNEL_LOOP(index, n) 
	{
		int temp = index;
		int Y_pos = temp % Y;
		int Z_pos = temp / Y;

		int current_pos = index * X;

		for (int X_pos = 0; X_pos < X; X_pos++, current_pos++)
		{
			int npos = X_pos * mX + Y_pos * mY + Z_pos * mZ;
			data_dst[npos] = data_src[current_pos] + alpha * data_dst[npos];
		}
	}
}

template <typename Dtype>
void cu_permute_3D_acc_gpu(const Dtype *data_src, Dtype *data_dst, const int X, const int Y, const int Z, const DimensionOrder transDirection, Dtype alpha)
{
	int num_kernels = Z * Y;

	//multiplier for X axis, Y axis, Z axis
	int mX, mY, mZ;
	switch (transDirection)
	{
	case XYZtoXYZ:
		mX = 1, mY = X, mZ = X * Y;
		break;
	case XYZtoXZY:
		mX = 1, mY = X * Z, mZ = X;
		break;
	case XYZtoYXZ:
		mX = Y, mY = 1, mZ = X * Y;
		break;
	case XYZtoYZX:
		mX = X * Y, mY = 1, mZ = Y;
		break;
	case XYZtoZXY:
		mX = Z, mY = X * Z, mZ = 1;
		break;
	case XYZtoZYX:
		mX = Y * Z, mY = Z, mZ = 1;
		break;
	}

	cu_trans_3D_acc_gpu_kernel<Dtype><<<CAFFE_GET_BLOCKS(num_kernels),
		CAFFE_CUDA_NUM_THREADS, 0, Caffe::get_current_cu_stream()>>>(
		num_kernels, data_src, data_dst, X, Y, Z, mX, mY, mZ, alpha);
}

template void cu_permute_3D_acc_gpu<float>(const float *data_src, float *data_dst, const int X, const int Y, const int Z, const DimensionOrder transDirection, float alpha);
template void cu_permute_3D_acc_gpu<double>(const double *data_src, double *data_dst, const int X, const int Y, const int Z, const DimensionOrder transDirection, double alpha);


template <typename Dtype>
__global__ void cu_trans_3D_carry_gpu_kernel(const int n, const Dtype* data_src, Dtype *data_dst,
											 const Dtype *data_carry,
    const int X, const int Y, const int Z, int mX, int mY, int mZ, Dtype alpha, Dtype beta)
{
	CUDA_KERNEL_LOOP(index, n) 
	{
		int temp = index;
		int Y_pos = temp % Y;
		int Z_pos = temp / Y;

		int current_pos = index * X;

		for (int X_pos = 0; X_pos < X; X_pos++, current_pos++)
		{
			int npos = X_pos * mX + Y_pos * mY + Z_pos * mZ;
			data_dst[npos] = data_src[current_pos] + alpha * data_dst[npos] + beta * data_carry[npos];
		}
	}
}

template <typename Dtype>
void cu_permute_3D_carry_gpu(const Dtype *data_src, Dtype *data_dst, Dtype * data_carry, const int X, const int Y, const int Z,
							 const DimensionOrder transDirection, Dtype alpha, Dtype beta)
{
	int num_kernels = Z * Y;

	//multiplier for X axis, Y axis, Z axis
	int mX, mY, mZ;
	switch (transDirection)
	{
	case XYZtoXYZ:
		mX = 1, mY = X, mZ = X * Y;
		break;
	case XYZtoXZY:
		mX = 1, mY = X * Z, mZ = X;
		break;
	case XYZtoYXZ:
		mX = Y, mY = 1, mZ = X * Y;
		break;
	case XYZtoYZX:
		mX = X * Y, mY = 1, mZ = Y;
		break;
	case XYZtoZXY:
		mX = Z, mY = X * Z, mZ = 1;
		break;
	case XYZtoZYX:
		mX = Y * Z, mY = Z, mZ = 1;
		break;
	}

	cu_trans_3D_carry_gpu_kernel<Dtype><<<CAFFE_GET_BLOCKS(num_kernels),
		CAFFE_CUDA_NUM_THREADS, 0, Caffe::get_current_cu_stream()>>>(
		num_kernels, data_src, data_dst, data_carry,  X, Y, Z, mX, mY, mZ, alpha, beta);
}

template void cu_permute_3D_carry_gpu<float>(const float *data_src, float *data_dst, float * data_carry, const int X, const int Y, const int Z, const DimensionOrder transDirection, float alpha, float beta);
template void cu_permute_3D_carry_gpu<double>(const double *data_src, double *data_dst, double * data_carry, const int X, const int Y, const int Z, const DimensionOrder transDirection, double alpha, double beta);


}  // namespace caffe
