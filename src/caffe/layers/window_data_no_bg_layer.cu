// Copyright 2014 BVLC and contributors.
//
// Based on data_layer.cpp by Yangqing Jia.

#include <stdint.h>
#include <pthread.h>

#include <string>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/io.hpp"
#include "caffe/vision_layers.hpp"

using std::string;
using std::map;
using std::pair;

// caffe.proto > LayerParameter > WindowDataParameter
//   'source' field specifies the window_file
//   'crop_size' indicates the desired warped size

namespace caffe {

INSTANTIATE_CLASS(WindowDataNoBgLayer);

}  // namespace caffe
